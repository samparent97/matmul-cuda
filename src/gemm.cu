#include "hip/hip_runtime.h"
#include "gemm.h"

#include "utils.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>

namespace swiftware::hpp {

__global__ void dotProduct(float *A, float *B, float *C, int m, int n, int k) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row = idx / n;
  int col = idx % n;
  float sum = 0.0f;

  if (row < m && col < n) {
    for (int l = 0; l < k; ++l) {
      sum += A[row * k + l] * B[l * n + col];
    }
  }
  C[idx] = sum;
}

__global__ void singleRow(float *A, float *B, float *C, int m, int n, int k) {
  // one thread = handles a whole row af A, therefore computes a whole row of C
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // not sharing memory, because threads don't reuse much within blocks
  // pre-tiling
  if (idx < m) {
    for (int j = 0; j < n; ++j) {
      float sum = 0.0f;
      for (int l = 0; l < k; ++l) {
        // if(idx == 1) {
        //   printf("A[%d][%d] = %f, B[%d][%d] = %f\n", idx, l, A[idx * k + l],
        //   l, j, B[l * n + j]);
        // }
        sum += A[idx * k + l] * B[l * n + j];
      }
      C[idx * n + j] = sum;
    }
  }

  //   if (idx < m) {
  //     for (int l = 0; l < k; ++l) {
  //       for (int j = 0; j < n; ++j) {
  //         C[idx * n + j] += A[idx * k + l] * B[l * n + j];
  //       }
  //   }
  // }
  // printf("Thread %d finished\n", idx);
}

float gemmGpuSingleRowDecomp(int m, int n, int k, const float *h_A,
                             const float *h_B, float *h_C, ScheduleParams Sp) {
  const size_t sizeA = m * k;
  const size_t sizeB = k * n;
  const size_t sizeC = m * n;

  // Allocate mem on GPU
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, sizeA * sizeof(float));
  hipMalloc((void **)&d_B, sizeB * sizeof(float));
  hipMalloc((void **)&d_C, sizeC * sizeof(float));

  // Move mem to gpu
  hipMemcpy(d_A, h_A, sizeA * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeB * sizeof(float), hipMemcpyHostToDevice);

  // figure out how many threadsPerBlock, size of blocks
  // dim3 block(Sp.TileSize1, 1, 1);
  // dim3 grid((m + Sp.TileSize1 - 1) / Sp.TileSize1, 1, 1);
  // Giving up on using dim3 for now, just give it ints
  int threadsPerBlock = 256;
  int blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;

  // hipDeviceSynchronize();
  // start timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // run kernel
  singleRow<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);

  // synchronize
  // hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // get timing measurement
  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);

  std::cout << "Time taken: " << elapsed << " ms" << std::endl;

  // get memory back
  hipMemcpy(h_C, d_C, sizeC * sizeof(float), hipMemcpyDeviceToHost);

  // cleanup memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  //   delete[] cpu_C;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed;
}

__global__ void twoDimTile(float *A, float *B, float *C, int m, int n, int k) {
  // TODO
}

float gemmGpuSingleElementDecomp(int m, int n, int k, const float *h_A,
                                 const float *h_B, float *h_C,
                                 ScheduleParams Sp) {
  const size_t sizeA = m * k;
  const size_t sizeB = k * n;
  const size_t sizeC = m * n;

  // Allocate mem on GPU
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, sizeA * sizeof(float));
  hipMalloc((void **)&d_B, sizeB * sizeof(float));
  hipMalloc((void **)&d_C, sizeC * sizeof(float));

  // Move mem to gpu
  hipMemcpy(d_A, h_A, sizeA * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeB * sizeof(float), hipMemcpyHostToDevice);

  // figure out how many threadsPerBlock, size of blocks
  dim3 block(1, 1, 1);
  dim3 grid((m * n + block.x - 1) / block.x, 1, 1);

  // start timing
  hipEvent_t start, stop;
  hipEventCreate(&start);

  // run kernel
  dotProduct<<<grid, block>>>(d_A, d_B, d_C, m, n, k);

  // synchronize
  hipDeviceSynchronize();
  hipEventCreate(&stop);

  // get timing measurement
  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);

  // get memory back
  hipMemcpy(h_C, d_C, sizeC * sizeof(float), hipMemcpyDeviceToHost);

  // Verify result against CPU calculation
  //   float *cpu_C = new float[sizeC];
  //   const float tol = 1e-5;
  //   bool correct = true;

  //   // CPU matrix multiplication for verification
  //   for (int i = 0; i < m; i++) {
  //     for (int j = 0; j < n; j++) {
  //       float sum = 0.0f;
  //       for (int l = 0; l < k; l++) {
  //         sum += h_A[i * k + l] * h_B[l * n + j];
  //       }
  //       cpu_C[i * n + j] = sum;

  //       // Compare GPU and CPU results
  //       if (std::abs(cpu_C[i * n + j] - h_C[i * n + j]) > tol) {
  //         correct = false;
  //         std::cout << "Mismatch at position (" << i << "," << j << "): ";
  //         std::cout << "CPU=" << cpu_C[i * n + j] << ", GPU=" << h_C[i * n +
  //         j]
  //                   << std::endl;
  //       }
  //     }
  //   }

  //   if (correct) {
  //     std::cout << "Matrix multiplication results match within tolerance!"
  //               << std::endl;
  //   }

  // cleanup memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  //   delete[] cpu_C;

  return elapsed;
}

template <int TileSize>
__global__ void oneDimTile(float *A, float *B, float *C, int m, int n, int k) {
  // Details regarding this thread
  int by = blockIdx.y;
  int bx = blockIdx.x;

  int ty = threadIdx.y;
  int tx = threadIdx.x;

  // Working on C[i,j]
  int i = TileSize * by + ty;
  int j = TileSize * bx + tx;

  // Allocating shared memory
  __shared__ float sh_A[TileSize][TileSize];
  __shared__ float sh_B[TileSize][TileSize];

  // Parallel mat mul
  float value = 0;
  for (int tileCount = 0; tileCount < ceil((float)k / TileSize); tileCount++) {
    // Load Tiles into shared memory
    if ((i < m) && ((tileCount * TileSize + tx) < k))
      sh_A[ty][tx] = A[(i)*k + tileCount * TileSize + tx];
    else
      sh_A[ty][tx] = 0.0f;

    if (((tileCount * TileSize + ty) < k) && (j < n))
      sh_B[ty][tx] = B[(tileCount * TileSize + ty) * n + j];
    else
      sh_B[ty][tx] = 0.0f;
    __syncthreads();

    // Dot product
    for (int k = 0; k < TileSize; k++)
      value += sh_A[ty][k] * sh_B[k][tx];
    __syncthreads();
  }
  // Assigning calculated value
  if ((i < m) && (j < n))
    C[i * n + j] = value;
}

float gemmGpuOneDimTile(int m, int n, int k, const float *h_A, const float *h_B,
                        float *h_C, ScheduleParams Sp) {
  const size_t sizeA = m * k;
  const size_t sizeB = k * n;
  const size_t sizeC = m * n;

  // Allocate mem on GPU
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, sizeA * sizeof(float));
  hipMalloc((void **)&d_B, sizeB * sizeof(float));
  hipMalloc((void **)&d_C, sizeC * sizeof(float));

  // Move mem to gpu
  hipMemcpy(d_A, h_A, sizeA * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeB * sizeof(float), hipMemcpyHostToDevice);

  constexpr int TileSize = 32;
  // Kernel execution
  dim3 dim_block(TileSize, TileSize, 1);
  dim3 dim_grid(ceil(k / (float)(TileSize)), ceil(m / (float)(TileSize)), 1);

  // start timing
  hipEvent_t start, stop;
  hipEventCreate(&start);

  // run kernel
  oneDimTile<TileSize><<<dim_grid, dim_block>>>(d_A, d_B, d_C, m, n, k);

  // synchronize
  hipDeviceSynchronize();
  hipEventCreate(&stop);

  // get timing measurement
  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);

  // get memory back
  hipMemcpy(h_C, d_C, sizeC * sizeof(float), hipMemcpyDeviceToHost);

  // Verify result against CPU calculation
  //   float *cpu_C = new float[sizeC];
  //   const float tol = 1e-5;
  //   bool correct = true;

  //   // CPU matrix multiplication for verification
  //   for (int i = 0; i < m; i++) {
  //     for (int j = 0; j < n; j++) {
  //       float sum = 0.0f;
  //       for (int l = 0; l < k; l++) {
  //         sum += h_A[i * k + l] * h_B[l * n + j];
  //       }
  //       cpu_C[i * n + j] = sum;

  //       // Compare GPU and CPU results
  //       if (std::abs(cpu_C[i * n + j] - h_C[i * n + j]) > tol) {
  //         correct = false;
  //         std::cout << "Mismatch at position (" << i << "," << j << "): ";
  //         std::cout << "CPU=" << cpu_C[i * n + j] << ", GPU=" << h_C[i * n +
  //         j]
  //                   << std::endl;
  //       }
  //     }
  //   }

  //   if (correct) {
  //     std::cout << "Matrix multiplication results match within tolerance!"
  //               << std::endl;
  //   }

  // cleanup memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  //   delete[] cpu_C;

  return elapsed;
}

} // namespace swiftware::hpp